/**
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "ans/GpuANSEncode.h"
#include "ans/GpuANSDecode.h"
#include "ans/GpuANSCodec.h"
#include <cmath>
#include <memory>
#include <vector>

namespace multibyte_ans {

void ansEncodeBatch(
    int precision,
    uint8_t* in,
    uint32_t inSize,
    uint8_t* out,
    uint32_t* outSize,
    hipStream_t stream) {

  ansEncode(
      precision,
      in,
      inSize,
      out,
      outSize,
      stream);
}

void ansDecodeBatch(
    int precision,
    uint8_t* in,
    uint8_t* out,
    hipStream_t stream) {

    ansDecode(
        precision,
        in,
        out,
        stream);
}
}