#include "hip/hip_runtime.h"
/**
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */
#include "ans/GpuANSEncode.h"
#include "ans/GpuANSDecode.h"
#include "ans/GpuANSCodec.h"
#include <cmath>
#include <memory>
#include <vector>

namespace multibyte_ans {

void ansEncodeBatch(
    uint32_t maxNumcompressedWords,
    uint32_t maxNumCompressedBlocks,
    uint4* table_dev,
    uint32_t* tempHistogram_dev,
    uint32_t uncoalescedBlockStride,
    uint8_t* compressedBlocks_dev,
    uint32_t* compressedWords_dev,
    uint32_t* compressedWordsPrefix_dev,
    uint32_t sizeRequired,
    uint8_t* tempPrefixSum_dev,
    int precision,
    uint8_t* in,
    uint32_t inSize,
    uint8_t* out,
    uint32_t* outSize,
    hipStream_t stream) {
        
    ansEncode(
        maxNumcompressedWords,
        maxNumCompressedBlocks,
        table_dev,
        tempHistogram_dev,
        uncoalescedBlockStride,
        compressedBlocks_dev,
        compressedWords_dev,
        compressedWordsPrefix_dev,
        sizeRequired,
        tempPrefixSum_dev,
        precision,
        in,
        inSize,
        out,
        outSize,
        stream);
}

void ansDecodeBatch(
    int precision,
    uint8_t* in,
    uint8_t* out,
    hipStream_t stream) {

    ansDecode(
        precision,
        in,
        out,
        stream);
}
}