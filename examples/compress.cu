#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <chrono>
#include "ans/GpuANSEncode.h"
#include "ans/GpuANSCodec.h"

using namespace multibyte_ans;

void compressFileWithANS(
		const std::string& inputFilePath,//输入数据文件路径
		const std::string& tempFilePath,//压缩后文件保存路径
        uint32_t& batchSize,//原本数据规模
		uint32_t& compressedSize,//压缩后数据大小
		int precision,//ANS的精度
		hipStream_t stream
		) {
    //读取输入文件
    std::ifstream inputFile(inputFilePath, std::ios::binary | std::ios::ate);
    std::streamsize fileSize = inputFile.tellg();
    std::vector<uint8_t> fileData(fileSize);
    inputFile.seekg(0, std::ios::beg);
    inputFile.read(reinterpret_cast<char*>(fileData.data()), fileSize);//全部按照uint8_t读入
    inputFile.close();

    //传输输入文件的数据
    uint8_t* inPtrs;
    hipMalloc(&inPtrs, sizeof(uint8_t)*(fileSize));
    hipMemcpy(inPtrs, fileData.data(), fileSize*sizeof(uint8_t), hipMemcpyHostToDevice);

    //设置batchSize，只有一个batch
    batchSize = fileSize;

    //分配存储压缩后数据大小的GPU空间
    uint32_t* outCompressedSize;
    hipMalloc(&outCompressedSize, sizeof(uint32_t));

    //分配存储压缩后数据的GPU空间
    uint8_t* encPtrs;
    hipMalloc(&encPtrs, static_cast<uint64_t>(getMaxCompressedSize(fileSize)));

    uint32_t maxUncompressedWords = batchSize / sizeof(ANSDecodedT);
    uint32_t maxNumCompressedBlocks =
      (maxUncompressedWords + kDefaultBlockSize - 1) / kDefaultBlockSize;//一个batch的数据以kDefaultBlockSize作为基准划分数据，形成多个数据块

    uint4* table_dev;
    CUDA_VERIFY(hipMalloc(&table_dev, sizeof(uint4) * kNumSymbols));

    uint32_t* tempHistogram_dev;
    CUDA_VERIFY(hipMalloc(&tempHistogram_dev, sizeof(uint32_t) * kNumSymbols));

    uint32_t uncoalescedBlockStride =
      getMaxBlockSizeUnCoalesced(kDefaultBlockSize);

    uint8_t* compressedBlocks_dev;
    CUDA_VERIFY(hipMalloc(&compressedBlocks_dev, sizeof(uint8_t) * maxNumCompressedBlocks * uncoalescedBlockStride));

    uint32_t* compressedWords_dev;
    CUDA_VERIFY(hipMalloc(&compressedWords_dev, sizeof(uint32_t) * maxNumCompressedBlocks));

    uint32_t* compressedWordsPrefix_dev;
    CUDA_VERIFY(hipMalloc(&compressedWordsPrefix_dev, sizeof(uint32_t) * maxNumCompressedBlocks));

    auto sizeRequired =
        getBatchExclusivePrefixSumTempSize(
          maxNumCompressedBlocks);

    uint8_t* tempPrefixSum_dev = nullptr;
    CUDA_VERIFY(hipMalloc(&tempPrefixSum_dev, sizeof(uint8_t) * sizeRequired));

    std::cout<<"encode start!"<<std::endl;
    //计时
    double time = 0.0;
    for(int i = 0; i < 11; i ++){
    auto start = std::chrono::high_resolution_clock::now();  

    //压缩开始 
    ansEncode(
        maxUncompressedWords,
        maxNumCompressedBlocks,
        table_dev,
        tempHistogram_dev,
        uncoalescedBlockStride,
        compressedBlocks_dev,
        compressedWords_dev,
        compressedWordsPrefix_dev,
        sizeRequired,
        tempPrefixSum_dev,
        precision,
        inPtrs,//已经在dev
        batchSize,
        encPtrs,//GPU
        outCompressedSize,//GPU
        stream);
    hipStreamSynchronize(stream);

    auto end = std::chrono::high_resolution_clock::now();
    double comp_time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e3;  
    if(i > 5)
    {
        time += comp_time;
    }
    }
    //计算速度
    double c_bw = ( 1.0 * fileSize / 1e9 ) / ( (time / 5.0) * 1e-3 );  
    //输出结果
    std::cout << "comp   time " << std::fixed << std::setprecision(6) << (time / 5.0) << " ms B/W "   
                  << std::fixed << std::setprecision(1) << c_bw << " GB/s " << std::endl;
    
    //获取压缩后的数据大小
    uint32_t outsize;
    hipMemcpy(&outsize, outCompressedSize, sizeof(uint32_t), hipMemcpyDeviceToHost);
    compressedSize = outsize;
    //printf("compressed size %d\n", compressedSize);

    //保存压缩后的数据到tempFilePath
    std::ofstream outputFile(tempFilePath, std::ios::binary);
    std::vector<uint8_t> compressedData(outsize);
    hipMemcpy(compressedData.data(), encPtrs, outsize*sizeof(uint8_t), hipMemcpyDeviceToHost);
    outputFile.write(reinterpret_cast<const char*>(compressedData.data()), outsize*sizeof(uint8_t));
    outputFile.close();
}

// void decompressFileWithANS(
// 		const std::string& tempFilePath, //压缩文件路径
// 		const std::string& outputFilePath,   //解压缩后文件路径
//         uint32_t& batchSize,      //解压缩后的数据大小，原本数据大小      
//         uint32_t& compressedSize, //压缩后的数据大小          
// 		int precision,//精度
// 		hipStream_t stream) {
//     //读取压缩文件
//     std::ifstream inFile(tempFilePath, std::ios::binary);
//     std::vector<uint8_t> fileCompressedData(compressedSize);
//     inFile.read(reinterpret_cast<char*>(fileCompressedData.data()), compressedSize);
//     inFile.close();

//     uint8_t* filePtrs;//传输输入数据
//     hipMalloc(&filePtrs, sizeof(uint8_t)*(compressedSize));
//     hipMemcpy(filePtrs,fileCompressedData.data(),compressedSize*sizeof(uint8_t),hipMemcpyHostToDevice);

//     uint8_t* decPtrs;//分配保存解压缩数据的空间
//     hipMalloc(&decPtrs, sizeof(uint8_t)*(batchSize));
    
//     std::cout<<"decode start!"<<std::endl;
//     //计时
//     double decomp_time = 0.0;
//     auto start = std::chrono::high_resolution_clock::now();

//     //解压开始
//     ansDecode(
//         precision,//解压缩精度
//         filePtrs, //解压缩输入数据
//         decPtrs,//解压缩输出数据
//         stream);
//     hipStreamSynchronize(stream);
//     //printf("1\n");
//     auto end = std::chrono::high_resolution_clock::now();  
//     decomp_time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e3; 
    
//     //计算速度
//     double dc_bw = ( 1.0 * compressedSize / 1e9 ) / ( decomp_time * 1e-3 );
//     //输出结果
//     std::cout << "decomp time " << std::fixed << std::setprecision(3) << decomp_time << " ms B/W "   
//                   << std::fixed << std::setprecision(1) << dc_bw << " GB/s" << std::endl;
//     //保存解压后的文件到outputFilePath
//     std::ofstream outFile(outputFilePath, std::ios::binary);
//     std::vector<uint8_t> unCompressData(batchSize);
//     hipMemcpy(unCompressData.data(),decPtrs,batchSize*sizeof(uint8_t),hipMemcpyDeviceToHost);
//     outFile.write(reinterpret_cast<const char*>(unCompressData.data()), batchSize*sizeof(uint8_t));
//     outFile.close();
// }

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <inputfile> <tempfile> " << std::endl;
        return 1;
    }
    hipStream_t stream;   
    hipStreamCreate(&stream);
    uint32_t batchSize;
    uint32_t compressedSize;
    int precision = 10; 
    compressFileWithANS(
        argv[1], argv[2],
        batchSize,//压缩前数据的大小
        compressedSize,//压缩后的数据大小
        precision,//ANS的精度
        stream);
    printf("compress ratio: %f\n", 1.0 * batchSize / compressedSize);
	// decompressFileWithANS(
    //     argv[3],argv[4],
    //     batchSize,//原本的数据规模s
    //     compressedSize,//压缩后数据规模
    //     precision,//精度
    //     stream);
    std::cout << "Compression completed successfully." << std::endl;
    return 0;
}
