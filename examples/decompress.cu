#include <iostream>
#include <fstream>
#include <iomanip>
#include <chrono>
#include "ans/GpuANSDecode.h"
#include "ans/GpuANSCodec.h"

using namespace multibyte_ans;

void decompressFileWithANS(
		const std::string& tempFilePath, //压缩文件路径
		const std::string& outputFilePath,   //解压缩后文件路径
        uint32_t& batchSize,      //解压缩后的数据大小，原本数据大小      
        uint32_t& compressedSize, //压缩后的数据大小          
		int precision,//精度
		hipStream_t stream) {
    std::ifstream inFile0(tempFilePath, std::ios::binary);
    std::vector<uint8_t> fileCompressedHead(32);
    inFile0.read(reinterpret_cast<char*>(fileCompressedHead.data()), 32);
    auto Header = (ANSCoalescedHeader*)fileCompressedHead.data();
    compressedSize = Header->getTotalCompressedSize();
    batchSize = Header->getTotalUncompressedWords();
    // printf("batchSize: %d\n", batchSize);
    inFile0.close();
    // printf("totalCompressedSize: %d\n", totalCompressedSize);
    std::ifstream inFile1(tempFilePath, std::ios::binary);
    std::vector<uint8_t> fileCompressedData(compressedSize);
    inFile1.read(reinterpret_cast<char*>(fileCompressedData.data()), compressedSize);
    inFile1.close();
    uint8_t* filePtrs;
    hipMalloc(&filePtrs, sizeof(uint8_t)*(compressedSize));
    hipMemcpy(filePtrs,fileCompressedData.data(),compressedSize*sizeof(uint8_t),hipMemcpyHostToDevice);

    uint8_t* decPtrs;
    hipMalloc(&decPtrs, sizeof(uint8_t)*(batchSize));
    
    std::cout<<"decode start!"<<std::endl;
    double decomp_time = 0.0;
    double time = 0.0;
    for(int i = 0; i < 11; i ++){
    auto start = std::chrono::high_resolution_clock::now();

    //解压开始
    ansDecode(
        precision,//解压缩精度
        filePtrs, //解压缩输入数据
        decPtrs,//解压缩输出数据
        stream);
    hipStreamSynchronize(stream);
    //printf("1\n");
    auto end = std::chrono::high_resolution_clock::now();  
    decomp_time = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e3; 
    if(i > 5)
    {
        time += decomp_time;
    }
    }
    
    //计算速度
    double dc_bw = ( 1.0 * compressedSize / 1e9 ) / ( (time / 5.0) * 1e-3 );
    //输出结果
    std::cout << "decomp time " << std::fixed << std::setprecision(3) << (time / 5.0) << " ms B/W "   
                  << std::fixed << std::setprecision(1) << dc_bw << " GB/s" << std::endl;
    //保存解压后的文件到outputFilePath
    std::ofstream outFile(outputFilePath, std::ios::binary);
    std::vector<uint8_t> unCompressData(batchSize);
    hipMemcpy(unCompressData.data(),decPtrs,batchSize*sizeof(uint8_t),hipMemcpyDeviceToHost);
    outFile.write(reinterpret_cast<const char*>(unCompressData.data()), batchSize*sizeof(uint8_t));
    outFile.close();
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <input.file> <output.file>" << std::endl;
        return 1;
    }
    hipStream_t stream;   
    hipStreamCreate(&stream);
    uint32_t batchSize;
    uint32_t compressedSize;
    int precision = 10; 
	decompressFileWithANS(
        argv[1],argv[2],
        batchSize,//原本的数据规模
        compressedSize,//压缩后数据规模
        precision,//精度
        stream);
    std::cout << "Decompression completed successfully." << std::endl;
    return 0;
}
